#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

float *d_img, *d_ans, *d_kernel;
size_t img_pitch, ans_pitch, kernel_pitch;

__global__ void conv(float *d_ans, float *d_img, float *d_kernel,
                    size_t ans_pitch, size_t img_pitch, size_t kernel_pitch,
                    int width, int height, int k_size, int pad) {
    
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r >= height || c >= width)
        return;

    float res = 0.0;
    for (int kr = -pad; kr <= pad; kr++) {
        for (int kc = -pad; kc <= pad; kc++) {
            float img_rc = *((float*)((char*)d_img + (r + pad + kr) * img_pitch) + (c + pad + kc));
            float kernel_rc = *((float*)((char*)d_kernel + (kr + pad) * kernel_pitch) + (kc + pad));
            res += img_rc * kernel_rc;
        }
    }
    *((float*)((char*)d_ans + r * ans_pitch) + c) = res;
}

void mallocKernelAndAns(float *kernel_arr, int width, int height, int k_size, int pad) {

    hipMallocPitch((void **)&d_img, &img_pitch, (width + 2 * pad) * sizeof(float), height + 2 * pad);
    hipMallocPitch((void **)&d_ans, &ans_pitch, width * sizeof(float), height);
    hipMallocPitch((void **)&d_kernel, &kernel_pitch, k_size * sizeof(float), k_size);
    hipMemcpy2D(d_kernel, kernel_pitch, 
                    kernel_arr, k_size * sizeof(float), 
                    k_size * sizeof(float), k_size, 
                    hipMemcpyHostToDevice);

}

void convolution(float *img_arr, 
                 float *ans_arr,
                 int width, 
                 int height, 
                 int k_size, 
                 int pad) {

    // init cuda arr
    hipMemcpy2D(d_img, img_pitch, 
                img_arr, (width + 2 * pad) * sizeof(float), 
                (width + 2 * pad) * sizeof(float), (height + 2 * pad), 
                hipMemcpyHostToDevice);
    
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(width / BLOCK_SIZE + 1, height / BLOCK_SIZE + 1);
    conv<<<numBlock, blockSize>>>(d_ans, d_img, d_kernel, 
                                 ans_pitch, img_pitch, kernel_pitch,
                                 width, height, k_size, pad);
    
    hipMemcpy2D(ans_arr, width * sizeof(float), 
                 d_ans, ans_pitch, 
                 width * sizeof(float), height, 
                 hipMemcpyDeviceToHost);

}

void freeKernelAndAns() {
    hipFree(d_img);
    hipFree(d_ans);
    hipFree(d_kernel);
}