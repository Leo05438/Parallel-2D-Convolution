#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

float *d_img, *d_ans, *d_kernel;
size_t img_pitch, ans_pitch, kernel_pitch;

__global__ void conv(float *d_ans, float *d_img, float *d_kernel,
                    size_t ans_pitch, size_t img_pitch, size_t kernel_pitch,
                    int width, int height, int k_size, int pad) {

    int r = blockIdx.y * (blockDim.y - 2 * pad) + threadIdx.y;
    int c = blockIdx.x * (blockDim.x - 2 * pad) + threadIdx.x;

    if (r >= height + 2 * pad || c >= width + 2 * pad)
        return;
    
    __shared__ float shared_patch[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_kernel[BLOCK_SIZE][BLOCK_SIZE];

    // tiling : load patch and kernel to shared memory in a SM
    shared_patch[threadIdx.y][threadIdx.x] = 
        *((float*)((char*)d_img + r * img_pitch) + c);
    shared_kernel[threadIdx.y % k_size][threadIdx.x % k_size] = 
        *((float*)((char*)d_kernel + (threadIdx.y % k_size) * kernel_pitch) + threadIdx.x % k_size);
    __syncthreads();

    if (threadIdx.y < pad || threadIdx.y >= BLOCK_SIZE - pad || 
        threadIdx.x < pad || threadIdx.x >= BLOCK_SIZE - pad ||
        r >= height + pad || c >= width + pad)
        return;

    int ans_r = r - pad; 
    int ans_c = c - pad; 
    
    float res = 0.0;
    for (int kr = -pad; kr <= pad; kr++) {
        for (int kc = -pad; kc <= pad; kc++) {
            res += shared_patch[threadIdx.y + kr][threadIdx.x + kc] * shared_kernel[kr + pad][kc + pad];
        }
    }

    *((float*)((char*)d_ans + ans_r * ans_pitch) + ans_c) = res;
}

void mallocKernelAndAns(float *kernel_arr, int width, int height, int k_size, int pad) {

    hipMallocPitch((void **)&d_img, &img_pitch, (width + 2 * pad) * sizeof(float), height + 2 * pad);
    hipMallocPitch((void **)&d_ans, &ans_pitch, width * sizeof(float), height);
    hipMallocPitch((void **)&d_kernel, &kernel_pitch, k_size * sizeof(float), k_size);
    hipMemcpy2D(d_kernel, kernel_pitch, 
                    kernel_arr, k_size * sizeof(float), 
                    k_size * sizeof(float), k_size, 
                    hipMemcpyHostToDevice);
}

void convolution(float *img_arr, 
                 float *ans_arr,
                 int width, 
                 int height, 
                 int k_size, 
                 int pad) {

    // init cuda arr
    hipMemcpy2D(d_img, img_pitch, 
                img_arr, (width + 2 * pad) * sizeof(float), 
                (width + 2 * pad) * sizeof(float), (height + 2 * pad), 
                hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(width / (BLOCK_SIZE - 2 * pad) + 1, height / (BLOCK_SIZE - 2 * pad) + 1);
    conv<<<numBlock, blockSize>>>(d_ans, d_img, d_kernel, 
                                 ans_pitch, img_pitch, kernel_pitch,
                                 width, height, k_size, pad);
    
    hipMemcpy2D(ans_arr, width * sizeof(float), 
                d_ans, ans_pitch, 
                width * sizeof(float), height, 
                hipMemcpyDeviceToHost);
    
}

void freeKernelAndAns() {
    hipFree(d_img);
    hipFree(d_ans);
    hipFree(d_kernel);
}