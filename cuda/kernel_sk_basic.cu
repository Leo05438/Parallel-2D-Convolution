#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

float *d_img, *d_ans, *d_kernel;

__global__ void conv0(float *d_ans, float *d_img, float *d_kernel_0,
                    int width, int height, int k_size, int pad) {
    
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r >= height || c >= width)
        return;

    float res = 0.0;
    for (int ki = -pad; ki <= pad; ki++) {
        if (r + ki >= 0 && r + ki < height)
            res += d_img[(r + ki) * width + c] * d_kernel_0[(ki + pad)];
    }
    if (r == height - 1 && c == 0) printf("res 0 : %f\n", res);
    d_ans[r * width + c] = res;
}

__global__ void conv1(float *d_ans, float *d_img, float *d_kernel_1,
                    int width, int height, int k_size, int pad) {
    
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r >= height || c >= width)
        return;
    
    float res = 0.0;
    for (int ki = -pad; ki <= pad; ki++) {
        if (c + ki >= 0 && c + ki < width)
            res += d_img[r * width + (c + ki)] * d_kernel_1[(ki + pad)];
    }
    if (r == height - 1 && c == 0) printf("res 1 : %f\n", res);
    d_ans[r * width + c] = res;
}

void mallocKernelAndAns(float *kernel_arr, int width, int height, int k_size, int pad) {

    hipMalloc((void **)&d_img, width * height * sizeof(float));
    hipMalloc((void **)&d_ans, width * height * sizeof(float));
    hipMalloc((void **)&d_kernel, k_size * k_size * sizeof(float));
    hipMemcpy(d_kernel, kernel_arr, k_size * k_size * sizeof(float), hipMemcpyHostToDevice);

}

void convolution(float *img_arr, 
                 float *ans_arr,
                 int width, 
                 int height, 
                 int k_size, 
                 int pad) {

    // init cuda arr
    hipMemcpy(d_img, img_arr, width * height * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(width / BLOCK_SIZE + 1, height / BLOCK_SIZE + 1);
    conv0<<<numBlock, blockSize>>>(d_ans, d_img, (d_kernel), width, height, k_size, pad);
    conv1<<<numBlock, blockSize>>>(d_img, d_ans, (d_kernel + k_size), width, height, k_size, pad);
    
    hipMemcpy(ans_arr, d_img, width * height * sizeof(float), hipMemcpyDeviceToHost);
}

void freeKernelAndAns() {
    hipFree(d_img);
    hipFree(d_ans);
    hipFree(d_kernel);
}